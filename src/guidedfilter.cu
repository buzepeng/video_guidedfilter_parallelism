#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/transform.h>
#include <thrust/execution_policy.h>

#include "guidedfilter.cuh"
#include "utils.cuh"

GuidedFilter::GuidedFilter(int w, int h){

    m = width * height * sizeof(uchar3);
    n = width * height * sizeof(float3);

    hipMalloc<uchar3>(&d_image_input, m);
    hipMalloc<uchar3>(&d_image_p, m);
    hipMalloc<uchar3>(&d_image_output, m);

    hipMalloc<float3>(&d_input, n);
    hipMalloc<float3>(&d_p, n);
    hipMalloc<float3>(&d_output, n);
    hipMalloc<float3>(&d_mean_I, n);
    hipMalloc<float3>(&d_mean_p, n);
    hipMalloc<float3>(&d_mean_Ip, n);
    hipMalloc<float3>(&d_mean_II, n);
    hipMalloc<float3>(&d_var_I, n);
    hipMalloc<float3>(&d_cov_Ip, n);
    hipMalloc<float3>(&d_a, n);
    hipMalloc<float3>(&d_b, n);
    hipMalloc<float3>(&d_mean_a, n);
    hipMalloc<float3>(&d_mean_b, n);
    hipMalloc<float3>(&d_tmp, n);
    hipMalloc<float3>(&d_tmp2, n);
}

GuidedFilter::~GuidedFilter(){
    hipFree(d_image_input);
    hipFree(d_image_output);
    hipFree(d_image_p);
    hipFree(d_input);
    hipFree(d_p);
    hipFree(d_output);
    hipFree(d_mean_I);
    hipFree(d_mean_p);
    hipFree(d_mean_Ip);
    hipFree(d_mean_II);
    hipFree(d_var_I);
    hipFree(d_cov_Ip);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_mean_a);
    hipFree(d_mean_b);
    hipFree(d_tmp);
    hipFree(d_tmp2);
}

void GuidedFilter::filter(uchar3* image_input, uchar3* image_output, uchar3* image_p, hipStream_t stream){
    int GRID_W = ceil(width /(float)TILE_W)+1;
    int GRID_H = ceil(height / (float)TILE_H)+1;
    int width = this->width;
    int height = this->height;

    const dim3 block(BLOCK_W, BLOCK_H);
    const dim3 grid(GRID_W, GRID_H);

    hipMemcpy(d_image_input, image_input, width * height * sizeof(uchar3), hipMemcpyHostToDevice);
    hipMemcpy(d_image_p, image_p, width * height * sizeof(uchar3), hipMemcpyHostToDevice);

    thrust::transform(thrust::device, d_image_input, d_image_input+width*height, d_input, uchar2float());
    thrust::transform(thrust::device, d_image_p, d_image_p+width*height, d_p, uchar2float());

    thrust::transform(thrust::device, d_input, d_input+width*height, d_p, d_tmp, [=]__device__(float3 pixel1, float3 pixel2){return pixel1*pixel2;});
    thrust::transform(thrust::device, d_input, d_input+width*height, d_tmp2, [=]__device__(float3 pixel){return pixel*pixel;});

    mean_kernel<<<grid, block>>>(d_input, d_p, d_output, d_mean_I, d_mean_p, d_mean_Ip,d_mean_II, d_var_I, d_cov_Ip, d_a, d_b, d_tmp, d_tmp2, d_mean_a,
    d_mean_b, width, height, EPS);
    hipDeviceSynchronize();

    output_kernel<<<grid, block>>>(d_input, d_p, d_output, d_a, d_b,d_mean_a, d_mean_b, width, height, EPS);
    hipDeviceSynchronize();

    thrust::transform(thrust::device, d_output, d_output+width*height, d_image_output, float2uchar());
    hipMemcpy(image_output, d_image_output, width * height * sizeof(uchar3), hipMemcpyDeviceToHost);
}